#include "hip/hip_runtime.h"
/*
CUDA search binary tree and bilinear interpolation in the triangle v1.0:

Copyright (C) 2010 Brown University

This library is free software; you can redistribute it and/or
modify it under the terms of the GNU Lesser General Public
License as published by the Free Software Foundation; either
version 2.1 of the License, or (at your option) any later version.

This library is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
Lesser General Public License for more details.

You should have received a copy of the GNU Lesser General Public
License along with this library; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301
USA, or see <http://www.gnu.org/licenses/old-licenses/lgpl-2.1.html>.

Author: Zhu Xueyu
*/



#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <time.h>

#define CUDA_CHK(NAME, ARGS) { \
  hipError_t cuda_err_code = NAME ARGS; \
  if (cuda_err_code != hipSuccess) { \
    printf("%s failed with code %d\n", #NAME, cuda_err_code); \
    abort(); \
  } \
}

// using texture memory
texture<float, 1, hipReadModeElementType> texRef;
texture<float, 1, hipReadModeElementType> texRef2;
texture<float, 1, hipReadModeElementType> valx_t;
texture<float, 1, hipReadModeElementType> valy_t;
//const int max_threads = 512;
// input generation

#define MULT 1103515245
#define ADD 12345
#define MASK 0x7FFFFFFF
#define TWOTO31 2147483648.0

static int A = 1;
static int B = 0;
static int randx = 1;
static int lastrand;


static void drndset(int seed)
{
   A = 1;
   B = 0;
   randx = (A * seed + B) & MASK;
   A = (MULT * A) & MASK;
   B = (MULT * B + ADD) & MASK;
}


static double drnd()
{
   lastrand = randx;
   randx = (A * randx + B) & MASK;
   return (double)lastrand / TWOTO31;
}
    
//__constant__ int* x_ming, *x_maxg, *y_ming, *y_maxg;

__global__ void search_kernel(int len,int N, float *value_x, float* value_y, int *index, int *level_list_d, int* leaf_list_d, float* centerx_list_d, float *centery_list_d){
    
    int i;
    float xmin, ymin, xmax, ymax, width; 
    i = threadIdx.x + blockIdx.x * blockDim.x;
if (i<len){
    width = powf(2.0,-level_list_d[i]);
    xmin = centerx_list_d[i] - width;
    ymin = centery_list_d[i] - width;
    xmax = centerx_list_d[i] + width;
    ymax = centery_list_d[i] + width;
}
    const int s_width=512;
    __shared__ float x_loc[s_width], y_loc[s_width];
      int bound = max(N,len);
      for (int m = 0;m<N/s_width +1 ;m++)
      { 
        int k = m*s_width + threadIdx.x;
        if(k<N && threadIdx.x<s_width){
         x_loc[threadIdx.x] = value_x[m*s_width + threadIdx.x];
         y_loc[threadIdx.x] = value_y[m*s_width + threadIdx.x];
        }
        __syncthreads();  
 
        if (i< len){
           for (int j=0;j<s_width ; j++){
           if (x_loc[j] >= xmin && x_loc[j]<=xmax &&
               y_loc[j] > ymin && y_loc[j]<=ymax )
              index[j+m*s_width] = i;    
             //index[j] = leaf_list_d[i];    
            }

        }
        __syncthreads();   
      }
}


void search_cpu(int len,int N, float *value_x, float* value_y, int *index_cpu, int *level_list, int* leaf_list, float* centerx_list, float *centery_list){
       for (int i = 0; i< len; i++){
           float width = pow(2.0,-level_list[i]);   
           float xmin = centerx_list[i] - width;
           float ymin = centery_list[i] - width;
           float xmax = centerx_list[i] + width;
           float ymax = centery_list[i] + width;
          for (int j=0;j<N;j++){
           if (value_x[j] >= xmin && value_x[j]<=xmax &&
            value_y[j]>ymin && value_y[j]<=ymax)
              //index_cpu[j] = leaf_list[i];
              index_cpu[j] = i ;
          }
       }
}
    
#if 1
__global__ void interpolation(int N, float* value_x, float *value_y, int* index_g, int *level_list_d, float *centerx_list_d, float* centery_list_d,  float *T1_list_d, float* T2_list_d, float * T3_list_d, float* T4_list_d,float* interp_value){

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i< N){
	    int j = index_g[i];
	    float width = powf(2.0,-level_list_d[j]);
#if 1
	    float xmin = centerx_list_d[j] - width;
	    float ymin = centery_list_d[j] - width;
	    float xmax = centerx_list_d[j] + width;
	    float ymax = centery_list_d[j] + width; 

#endif
#if 0
    float xmin =tex1Dfetch(texRef, i)  - width;
    float ymin =tex1Dfetch(texRef2, i)  - width;
    float xmax = tex1Dfetch(texRef, i) + width;
    float ymax = tex1Dfetch(texRef2, i) + width;
#endif
	    // rescale x,y in the local cell
	    float x_ref = (value_x[i]-xmin)/(xmax-xmin);
	    float y_ref = (value_y[i]-ymin)/(ymax-xmin);
	   
	    // pickup the interpolation triangle 
            float x_nodes[3], y_nodes[3], var[3];
	    x_nodes[0] = xmin;
	    x_nodes[1] = xmax ;
	    x_nodes[2] = x_ref>=y_ref?  xmax: xmin;
	    
            y_nodes[0] = ymin;
	    y_nodes[1] = x_ref>=y_ref? ymin:ymax ;
	    y_nodes[2] = ymax ;
	    
            var[0] = T1_list_d[j];
	    var[1] = x_ref>=y_ref? T2_list_d[j]: T3_list_d[j] ;
	    var[2] = x_ref>=y_ref? T3_list_d[j]: T4_list_d[j];
	float A = y_nodes[0]*(var[1]- var[2])  
                  +  y_nodes[1]*(var[2] - var[0]) 
                  +  y_nodes[2]*(var[0] - var[1]);

	float B = var[0]*(x_nodes[1] - x_nodes[2])
                   + var[1]*(x_nodes[2] - x_nodes[0])
                   +  var[2]*(x_nodes[0] - x_nodes[1]);

	float C = x_nodes[0]*(y_nodes[1] - y_nodes[2])
                  + x_nodes[1]*(y_nodes[2] - y_nodes[0])
                  + x_nodes[2]*(y_nodes[0] - y_nodes[1]);

	float D = -A*x_nodes[0] - B*y_nodes[0] - C*var[0];
	interp_value[i] = -(A*value_x[i] + B*value_y[i] + D)/C;
   }
}
#endif

void interpolation_cpu(int N, float* value_x, float *value_y, int* index_g, int *level_list_d, float *centerx_list_d, float* centery_list_d,  float *T1_list_d, float* T2_list_d, float * T3_list_d, float* T4_list_d,float* interp_value){
    std::cout << "interpolation on cpu!"<<std::endl;
    for(int i = 0;i< N;i++){
	    int j = index_g[i];
	    float width = powf(2.0,-level_list_d[j]);
	    float xmin = centerx_list_d[j] - width;
	    float ymin = centery_list_d[j] - width;
	    float xmax = centerx_list_d[j] + width;
	    float ymax = centery_list_d[j] + width; 

	    // rescale x,y in the local cell
	    float x_ref = (value_x[i]-xmin)/(xmax-xmin);
	    float y_ref = (value_y[i]-ymin)/(ymax-xmin);
	   
	    // pickup the interpolation triangle 
	    float x_nodes[3], y_nodes[3], var[3];
	    x_nodes[0] = xmin;
	    x_nodes[1] = x_ref>=y_ref?  xmax: xmax ;
	    x_nodes[2] = x_ref>=y_ref?  xmax: xmin;

	    y_nodes[0] = ymin;
	    y_nodes[1] = x_ref>=y_ref? ymin:ymax ;
	    y_nodes[2] = x_ref>=y_ref? ymax:ymax ;
	   
	    var[0] = T1_list_d[j];
	    var[1] = x_ref>=y_ref? T2_list_d[j]: T3_list_d[j] ;
	    var[2] = x_ref>=y_ref? T3_list_d[j]: T4_list_d[j];

	float A = y_nodes[0]*(var[1]- var[2])  +  y_nodes[1]*(var[2] - var[0]) +  y_nodes[2]*(var[0] - var[1]);

	float B = var[0]*(x_nodes[1] - x_nodes[2]) + var[1]*(x_nodes[2] - x_nodes[0]) +  var[2]*(x_nodes[0] - x_nodes[1]);

	float C = x_nodes[0]*(y_nodes[1] - y_nodes[2]) + x_nodes[1]*(y_nodes[2] - y_nodes[0]) + x_nodes[2]*(y_nodes[0] - y_nodes[1]);

	float D = -A*x_nodes[0] - B*y_nodes[0] - C*var[0];
	interp_value[i] = -(A*value_x[i] + B*value_y[i] + D)/C;

   }
}

int main( int argc, char** argv)
{
    // ----------------------v-------------------------------------------------
    CUDA_CHK(hipSetDevice, (3)); // EDIT ME!
    // ----------------------^-------------------------------------------------
   
        // Read the database
        using namespace std;
	int num_nodes, num_leafs;
	float rootwidth, xmin, xmax, ymin, ymax;
	int *level_list, *leaf_list;
        float *centerx_list, *centery_list;
	float *T1_list, *T2_list, *T3_list, *T4_list,*P1_list, *P2_list
	,*P3_list, *P4_list; // variable lists
        
        ifstream myfile("RPTBDB.dat");
        myfile >> num_nodes;
        myfile >> ymin >> ymax >> xmin >> xmax;
        myfile >> num_leafs >> rootwidth >> rootwidth;
        
        unsigned int bytes; 
        int fbytes = num_leafs*sizeof(float);
 	bytes = num_leafs * sizeof(int);
	int dbytes = sizeof(float);
	
        level_list = (int *) malloc( bytes);
	leaf_list = (int  *) malloc( bytes);
	centerx_list = (float *) malloc( fbytes);
	centery_list= (float *) malloc( fbytes);
	T1_list = (float *) malloc( fbytes);
	T2_list = (float *) malloc( fbytes);
	T3_list= (float *) malloc( fbytes);
	T4_list= (float *) malloc( fbytes);
	P1_list = (float *) malloc( fbytes);
	P2_list = (float *) malloc( fbytes);
	P3_list= (float *) malloc( fbytes);
	P4_list= (float *) malloc( fbytes);
        if (myfile.is_open())
	{
	  for(int i=0;i< num_leafs; i++){
	     myfile >> level_list[i] >> leaf_list[i];
	     myfile >> centerx_list[i] >> centery_list[i];
	     myfile >> T1_list[i] >> P1_list[i];
	     myfile >> T2_list[i] >> P2_list[i];
	     myfile >> T3_list[i] >> P3_list[i];
	     myfile >> T4_list[i] >> P4_list[i];
             }
	}
	myfile.close();

	int size= num_leafs; // numbet of elements to reduce 

	// allocate variables on GPU
	int *level_list_d, *leaf_list_d;
        float *centerx_list_d, *centery_list_d;
	float *T1_list_d, *T2_list_d, *T3_list_d, *T4_list_d,*P1_list_d, *P2_list_d;
        int *index, *index_g =NULL;
        float *value_x, *value_y, *value_x_d, *value_y_d, *interp_h, *interp_d, *interp;
       
        //rescale the input the data 
        int N=100*1000;
        value_x = (float *) malloc( N*dbytes);
        value_y = (float *) malloc( N*dbytes);
        index = (int *) malloc( N*sizeof(int));
        interp_h = (float *) malloc( N*dbytes);
        interp = (float *) malloc( N*dbytes);

        drndset(9);
        int *index_cpu;
        index_cpu = (int *) malloc( N*sizeof(int));

        for (int i=0; i < N; i++){
		value_x[i] = drnd()*600 + 400;
		value_y[i] = drnd()*2.0 - 1.0;
		value_x[i] = (value_x[i]-xmin)/(xmax-xmin);
		value_y[i] = (value_y[i]-ymin)/(ymax-ymin);
                index[i] = -1;
                index_cpu[i]=-1;
                interp[i] = -1;
                interp_h[i] = -1;
                //cout << i << " " <<value_x[i] << " " << value_y[i]<<endl;
        }
    
    clock_t starttime, endtime; 
    starttime = clock();
    search_cpu(size, N, value_x, value_y, index_cpu, level_list, leaf_list, centerx_list, centery_list);
    interpolation_cpu(N, value_x, value_y, index_cpu, level_list,centerx_list, centery_list, T1_list, T2_list, T3_list, T4_list,interp);
    endtime = clock();
    
    // allocate device memory and data
    cout << "allocating memory on GPU!" << endl;
    CUDA_CHK(hipMalloc, ((void**) &level_list_d, size*sizeof(int)));
    CUDA_CHK(hipMalloc, ((void**) &leaf_list_d, size*sizeof(int)));
    CUDA_CHK(hipMalloc, ((void**) &centerx_list_d, size*sizeof(float)));
    CUDA_CHK(hipMalloc, ((void**) &centery_list_d, size*sizeof(float)));
    
    CUDA_CHK(hipMalloc, ((void**) &value_x_d, N*sizeof(float)));
    CUDA_CHK(hipMalloc, ((void**) &value_y_d, N*sizeof(float)));
    CUDA_CHK(hipMalloc, ((void**) &index_g, N*sizeof(int)));
#if 1    
    CUDA_CHK(hipBindTexture, (0,texRef,centerx_list_d, size*sizeof(float)));
    CUDA_CHK(hipBindTexture, (0,texRef2,centery_list_d, size*sizeof(float)));
    CUDA_CHK(hipBindTexture, (0,valx_t,value_x_d, N*sizeof(float)));
    CUDA_CHK(hipBindTexture, (0,valy_t,value_y_d, N*sizeof(float)));
#endif
#if 1 
    CUDA_CHK(hipMalloc, ((void**) &T1_list_d, fbytes));
    CUDA_CHK(hipMalloc, ((void**) &T2_list_d, fbytes));
    CUDA_CHK(hipMalloc, ((void**) &T3_list_d, fbytes));
    CUDA_CHK(hipMalloc, ((void**) &T4_list_d, fbytes));
    CUDA_CHK(hipMalloc, ((void**) &interp_d, N*dbytes));
#endif
    cout << "transfering data to GPU!" << endl;
    CUDA_CHK(hipMemcpy, (level_list_d,level_list, size*sizeof(int),
                hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (leaf_list_d,leaf_list, size*sizeof(int),
                hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (centerx_list_d,centerx_list, size*sizeof(float),
                hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (centery_list_d,centery_list, size*sizeof(float),
                hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (value_x_d,value_x,N*sizeof(float),
                hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (value_y_d,value_y,N*sizeof(float),
                hipMemcpyHostToDevice));
 #if 1  
    CUDA_CHK(hipMemcpy, (T1_list_d, T1_list, fbytes,
                hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (T2_list_d, T2_list, fbytes,
                hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (T3_list_d, T3_list, fbytes,
               hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (T4_list_d, T4_list, fbytes,
                hipMemcpyHostToDevice));
    
    CUDA_CHK(hipMemcpy, (interp_d,interp_h,N*dbytes,
                hipMemcpyHostToDevice));
#endif
    CUDA_CHK(hipMemcpy, (index_g, index, N*sizeof(int),
                hipMemcpyHostToDevice));

    cout << "launching the kernel..." << endl;
    // run the kernel
    int num_threads = 512;
    int num_blocks = (size+ num_threads - 1)/num_threads ;
    cout << size << " "<<  num_threads << " " << num_blocks << " here" << endl;
    // measure the time
    
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);  hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // locate the cell
    search_kernel<<<num_blocks, num_threads>>>(size, N, value_x_d, value_y_d, index_g, level_list_d, leaf_list_d, centerx_list_d, centery_list_d); 

    printf("search kernel finished!\n");
    hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
    float search_time = time;
    num_blocks = (N+ num_threads - 1)/num_threads ;
    cout << size << " "<<  num_threads << " " << num_blocks << " here" << endl;
    interpolation<<<num_blocks, num_threads>>>(N, value_x_d, value_y_d, index_g, level_list_d,centerx_list_d, centery_list_d, T1_list_d, T2_list_d, T3_list_d, T4_list_d,interp_d);
   
    hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

    float interpolation_time = time-search_time;
    hipEventDestroy(start);  hipEventDestroy(stop);

    printf("interpolation kernel finished!\n");
    
    // copy back the result
    CUDA_CHK(hipMemcpy, (index,index_g, N*sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHK(hipMemcpy, (interp_h, interp_d, N*sizeof(float), hipMemcpyDeviceToHost));

    // check the result
    for (int i=0; i < N; i++){
       assert(interp[i]=interp_h[i]);
    if (index[i]<0)
       printf("cell %d is not in this range!, cpu: %d\n", i, index_cpu[i]);
    
    //else
       //printf("the value is cell : %d %d \n",index[i],index_cpu[i] ); 
    //   printf("the value is cell : %d %d %f %f\n",index[i],index_cpu[i], interp_h[i], interp[i] ); 
    }
  
    //output the time
     //printf("GPU: %.1f ms\n", time);
     printf("GPU: search_time: %10.5f ms, interpolation_time: %10.5f ms, total_time: %10.5f ms\n", search_time, interpolation_time, search_time + interpolation_time);
     printf("CPU %ld ms\n", (int) (1000.0f * (endtime - starttime) / CLOCKS_PER_SEC));

  
    //clean up
  CUDA_CHK(hipUnbindTexture,(texRef));
  CUDA_CHK(hipUnbindTexture,(texRef2));

	CUDA_CHK(hipFree, (level_list_d);  );
	CUDA_CHK(hipFree, (leaf_list_d);   );
	CUDA_CHK(hipFree, (centerx_list_d););
	CUDA_CHK(hipFree, (centery_list_d););
	CUDA_CHK(hipFree, (value_x_d));
	CUDA_CHK(hipFree, (value_y_d));
	CUDA_CHK(hipFree, (index_g));
	
        free(level_list);
	free(leaf_list);
	free(centerx_list);
	free(centery_list);
	free(value_x);
	free(value_y);
}
