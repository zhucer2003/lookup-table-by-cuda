
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <sstream>
#define CUDA_CHK(NAME, ARGS) { \
  hipError_t cuda_err_code = NAME ARGS; \
  if (cuda_err_code != hipSuccess) { \
    printf("%s failed with code %d\n", #NAME, cuda_err_code); \
    abort(); \
  } \
}

//const int max_threads = 512;
// input generation

#define MULT 1103515245
#define ADD 12345
#define MASK 0x7FFFFFFF
#define TWOTO31 2147483648.0

static int A = 1;
static int B = 0;
static int randx = 1;
static int lastrand;


static void drndset(int seed)
{
   A = 1;
   B = 0;
   randx = (A * seed + B) & MASK;
   A = (MULT * A) & MASK;
   B = (MULT * B + ADD) & MASK;
}


static double drnd()
{
   lastrand = randx;
   randx = (A * randx + B) & MASK;
   return (double)lastrand / TWOTO31;
}
    
//__constant__ int* x_ming, *x_maxg, *y_ming, *y_maxg;

__global__ void search_kernel(int len,int N, float *value_x, float* value_y, int *index, int *level_list_d, int* leaf_list_d, float* centerx_list_d, float *centery_list_d){
    
    int i;
    
    i = threadIdx.x + blockIdx.x * blockDim.x;
    float width = powf(2.0,-level_list_d[i]);
    float xmin = centerx_list_d[i] - width;
    float ymin = centery_list_d[i] - width;
    float xmax = centerx_list_d[i] + width;
    float ymax = centery_list_d[i] + width;
    if (i<len){
      for (int j=0;j<N; j++){
        if (value_x[j] > xmin && value_x[j]<=xmax &&
            value_y[j] > ymin && value_y[j]<=ymax)
           index[j] = leaf_list_d[i];    
      }
    }
   
}
#if 1
__global__ void interpolation(int N, float* value_x, float *value_y, int* index_g, int *level_list_d, float *centerx_list_d, float* centery_list_d,  float *T1_list_d, float* T2_list_d, float * T3_list_d, float* T4_list_d,float* interp_value){

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i< N){
    int j = index_g[i];
    float width = powf(2.0,-level_list_d[j]);
    float xmin = centerx_list_d[j] - width;
    float ymin = centery_list_d[j] - width;
    float xmax = centerx_list_d[j] + width;
    float ymax = centery_list_d[j] + width; 

    // rescale x,y in the local cell
    float x_ref = (value_x[i]-xmin)/(xmax-xmin);
    float y_ref = (value_x[i]-xmin)/(xmax-xmin);
   
    // pickup the interpolation triangle 
    float x_nodes[3], y_nodes[3], var[3];
    x_nodes[0] = xmin;
    x_nodes[1] = x_ref>=y_ref?  xmax: xmax ;
    x_nodes[2] = x_ref>=y_ref?  xmax: xmin;

    y_nodes[0] = ymin;
    y_nodes[1] = x_ref>=y_ref? ymin:ymax ;
    y_nodes[2] = x_ref>=y_ref? ymax:ymax ;
   
    var[0] = T1_list_d[j];
    var[1] = x_ref>y_ref? T2_list_d[j]: T3_list_d[j] ;
    var[2] = x_ref>y_ref? T3_list_d[j]: T4_list_d[j];

   // demonstrate single variable
   float cof_z = ( x_nodes[1]- x_nodes[0] ) * ( y_nodes[2]- y_nodes[0] ) 
         - ( x_nodes[2]- x_nodes[0] ) * (y_nodes[1]- y_nodes[0] );
   
   float cof_y = (var[1] - var[0]) * ( x_nodes[2]- x_nodes[0] ) 
                 - (var[2] - var[1]) *( x_nodes[1]- x_nodes[0] ) ;

   float cof_x = (var[2]- var[1]) * (y_nodes[1]- y_nodes[0] )
                 - (var[1] - var[0]) *( y_nodes[2]- y_nodes[0] ) ;
   
   interp_value[i] = var[0] - ((ymin - y_ref) * cof_y +  ((xmin - x_ref))*cof_x )/cof_z ;
   }
}
#endif

int main( int argc, char** argv)
{
    // ----------------------v-------------------------------------------------
    CUDA_CHK(hipSetDevice, (3)); // EDIT ME!
    // ----------------------^-------------------------------------------------
    //cudaDeviceProp devProp;
    //cudaGetDeviceProperties ( &devProp, 2 );
   
        // Read the database
        using namespace std;
	int num_nodes, num_leafs;
	float rootwidth, xmin, xmax, ymin, ymax;
	int *level_list, *leaf_list;
        float *centerx_list, *centery_list;
	float *T1_list, *T2_list, *T3_list, *T4_list,*P1_list, *P2_list
	,*P3_list, *P4_list; // variable lists
        
        ifstream myfile("RPTBDB.dat");
        myfile >> num_nodes;
        myfile >> xmin >> xmax >> ymin >> ymax;
        myfile >> num_leafs >> rootwidth >> rootwidth;
        
        unsigned int bytes; 
        int fbytes = num_leafs*sizeof(float);
 	bytes = num_leafs * sizeof(int);
	int dbytes = sizeof(float);
	
        level_list = (int *) malloc( bytes);
	leaf_list = (int  *) malloc( bytes);
	centerx_list = (float *) malloc( fbytes);
	centery_list= (float *) malloc( fbytes);
	T1_list = (float *) malloc( fbytes);
	T2_list = (float *) malloc( fbytes);
	T3_list= (float *) malloc( fbytes);
	T4_list= (float *) malloc( fbytes);
	P1_list = (float *) malloc( fbytes);
	P2_list = (float *) malloc( fbytes);
	P3_list= (float *) malloc( fbytes);
	P4_list= (float *) malloc( fbytes);
        if (myfile.is_open())
	{
	  for(int i=0;i< num_leafs; i++){
	     myfile >> level_list[i] >> leaf_list[i];
	     myfile >> centerx_list[i] >> centery_list[i];
	     myfile >> T1_list[i] >> P1_list[i];
	     myfile >> T2_list[i] >> P2_list[i];
	     myfile >> T3_list[i] >> P3_list[i];
	     myfile >> T4_list[i] >> P4_list[i];
           }
	}
	myfile.close();

	int size= num_leafs; // numbet of elements to reduce 

	
    

	// allocate variables on GPU
	int *level_list_d, *leaf_list_d;
        float *centerx_list_d, *centery_list_d;
	float *T1_list_d, *T2_list_d, *T3_list_d, *T4_list_d,*P1_list_d, *P2_list_d;
        int *index, *index_g =NULL;
        float *value_x, *value_y, *value_x_d, *value_y_d, *interp_cpu, *interp_d;
       
        //rescale the input the data 
        int N= 100;
        value_x = (float *) malloc( N*dbytes);
        value_y = (float *) malloc( N*dbytes);
        index = (int *) malloc( N*sizeof(int));
        interp_cpu = (float *) malloc( N*dbytes);

        drndset(9);
        int index_cpu[100];
        for (int i=0; i < N; i++){
		value_x[i] = drnd()*2.0 - 1.0;
		value_y[i] = drnd()*600 +400;
		value_x[i] = (value_x[i]-xmin)/(xmax-xmin);
		value_y[i] = (value_y[i]-ymin)/(ymax-ymin);
                index[i] = -1;
                index_cpu[i]=-1;
                cout << i << " " <<value_x[i] << " " << value_y[i]<<endl;
        }
       
       for (int i = 0; i< size; i++){
           float width = pow(2.0,-level_list[i]);   
           xmin = centerx_list[i] - width;
           ymin = centery_list[i] - width;
           xmax = centerx_list[i] + width;
           ymax = centery_list[i] + width;
          for (int j=0;j<N;j++){
           if (value_x[j] > xmin && value_x[j]<=xmax &&
            value_y[j]>ymin && value_y[j]<=ymax)
              index_cpu[j] = leaf_list[i];
          }
       }    
    // allocate device memory and data
    cout << "allocating memory on GPU!" << endl;
    CUDA_CHK(hipMalloc, ((void**) &level_list_d, size*sizeof(int)));
    CUDA_CHK(hipMalloc, ((void**) &leaf_list_d, size*sizeof(int)));
    CUDA_CHK(hipMalloc, ((void**) &centerx_list_d, size*sizeof(float)));
    CUDA_CHK(hipMalloc, ((void**) &centery_list_d, size*sizeof(float)));
    CUDA_CHK(hipMalloc, ((void**) &value_x_d, N*sizeof(float)));
    CUDA_CHK(hipMalloc, ((void**) &value_y_d, N*sizeof(float)));
    CUDA_CHK(hipMalloc, ((void**) &index_g, N*sizeof(int)));
#if 1 
    CUDA_CHK(hipMalloc, ((void**) &T1_list_d, fbytes));
    CUDA_CHK(hipMalloc, ((void**) &T2_list_d, fbytes));
    CUDA_CHK(hipMalloc, ((void**) &T3_list_d, fbytes));
    CUDA_CHK(hipMalloc, ((void**) &T4_list_d, fbytes));
    CUDA_CHK(hipMalloc, ((void**) &interp_d, N*dbytes));
#endif
    cout << "transfering data to GPU!" << endl;
    CUDA_CHK(hipMemcpy, (level_list_d,level_list, size*sizeof(int),
                hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (leaf_list_d,leaf_list, size*sizeof(int),
                hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (centerx_list_d,centerx_list, size*sizeof(float),
                hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (centery_list_d,centery_list, size*sizeof(float),
                hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (value_x_d,value_x,N*sizeof(float),
                hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (value_y_d,value_y,N*sizeof(float),
                hipMemcpyHostToDevice));
 #if 1  
    CUDA_CHK(hipMemcpy, (T1_list_d, T1_list, fbytes,
                hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (T2_list_d, T2_list, fbytes,
                hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (T3_list_d, T3_list, fbytes,
               hipMemcpyHostToDevice));
    CUDA_CHK(hipMemcpy, (T4_list_d, T4_list, fbytes,
                hipMemcpyHostToDevice));
    
    CUDA_CHK(hipMemcpy, (interp_d,interp_cpu,N*dbytes,
                hipMemcpyHostToDevice));
#endif
    CUDA_CHK(hipMemcpy, (index_g, index, N*sizeof(int),
                hipMemcpyHostToDevice));

    cout << "launching the kernel..." << endl;
    // run the kernel
    int num_threads = 256;
    int num_blocks = size/256 + 1;
    
    // locate the cell
    search_kernel<<<num_blocks, num_threads>>>(size, N, value_x_d, value_y_d, index_g, level_list_d, leaf_list_d, centerx_list_d, centery_list_d); 

    printf("kernel finished!\n");

    interpolation<<<num_blocks, num_threads>>>(N, value_x_d, value_y_d, index_g, level_list_d,centerx_list_d, centery_list_d, T1_list_d, T2_list_d, T3_list_d, T4_list_d,interp_d);

    // copy back the result
    CUDA_CHK(hipMemcpy, (index,index_g, N*sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHK(hipMemcpy, (interp_cpu,interp_d, N*sizeof(float), hipMemcpyDeviceToHost));

    // check the result
    for (int i=0; i < N; i++){
    if (index[i]<0)
       printf("cell %d is not in this range!, cpu: %d\n", i, index_cpu[i]);
    else
       //printf("the value is cell : %d %d \n",index[i],index_cpu[i] ); 
       printf("the value is cell : %d %d %f\n",index[i],index_cpu[i], interp_cpu[i] ); 
    }
    //clean up
	CUDA_CHK(hipFree, (level_list_d);  );
	CUDA_CHK(hipFree, (leaf_list_d);   );
	CUDA_CHK(hipFree, (centerx_list_d););
	CUDA_CHK(hipFree, (centery_list_d););
	CUDA_CHK(hipFree, (value_x_d));
	CUDA_CHK(hipFree, (value_y_d));
	CUDA_CHK(hipFree, (index_g));
	
        free(level_list);
	free(leaf_list);
	free(centerx_list);
	free(centery_list);
	free(value_x);
	free(value_y);
}
